
#include <hip/hip_runtime.h>
struct Conv2DOp {
    size_t stride;
    size_t padding;
    size_t kernel;
    size_t batch;
    size_t chan_in;
    size_t chan_out;
    size_t h_in;
    size_t h_out;
    size_t w_in;
    size_t w_out;
};

extern "C" __global__ void unfold_input_into_patches(
    const Conv2DOp op,
    const float *image, // 4d (Batch, Channels, Height, Width)
    float *patches // 6d (Batch, Channels, KernelSize, KernelSize, HeightOut, WidthOut)
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    const auto patches_numel = op.batch * op.chan_in * op.kernel * op.kernel * op.h_out * op.w_out;
    if (i >= patches_numel) {
        return;
    }

    // patches shape is (B, C, K, K, h_out, w_out)
    unsigned int idx = i;
    const size_t ow = idx % op.w_out;
    idx /= op.w_out;
    const size_t oh = idx % op.h_out;
    idx /= op.h_out;
    const size_t k2 = idx % op.kernel;
    idx /= op.kernel;
    const size_t k1 = idx % op.kernel;
    idx /= op.kernel;
    const size_t c = idx % op.chan_in;
    idx /= op.chan_in;
    const size_t b = idx % op.batch;
    idx /= op.batch;

    const size_t y_plus_p = oh * op.stride + k1;
    if (y_plus_p < op.padding) {
        return;
    }
    const size_t y = y_plus_p - op.padding;
    if (y >= op.h_in) {
        return;
    }

    const size_t x_plus_p = ow * op.stride + k2;
    if (x_plus_p < op.padding) {
        return;
    }
    const size_t x = x_plus_p - op.padding;
    if (x >= op.w_in) {
        return;
    }

    const size_t i_image = b * (op.chan_in * op.h_in * op.w_in) + c * (op.h_in * op.w_in) + y * (op.w_in) + x;
    patches[i] = image[i_image];
}

extern "C" __global__ void unfold_output_into_patches(
    const Conv2DOp op,
    const float *image_out, // 4d (Batch, ChanOut, HeightOut, WidthOut)
    float *patches // 6d (Batch, ChanOut, KernelSize, KernelSize, HeightIn, WidthIn)
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    const auto patches_numel = op.batch * op.chan_out * op.kernel * op.kernel * op.h_in * op.w_in;
    if (i >= patches_numel) {
        return;
    }

    unsigned int idx = i;
    const size_t x = idx % op.w_in;
    idx /= op.w_in;
    const size_t y = idx % op.h_in;
    idx /= op.h_in;
    const size_t k2 = idx % op.kernel;
    idx /= op.kernel;
    const size_t k1 = idx % op.kernel;
    idx /= op.kernel;
    const size_t o = idx % op.chan_out;
    idx /= op.chan_out;
    const size_t b = idx % op.batch;
    idx /= op.batch;

    size_t oh = y + op.padding;
    if (oh < k1) {
        return;
    }
    oh -= k1;
    if (oh % op.stride != 0) {
        return;
    }
    oh /= op.stride;
    if (oh >= op.h_out) {
        return;
    }
    
    size_t ow = x + op.padding;
    if (ow < k2) {
        return;
    }
    ow -= k2;
    if (ow % op.stride != 0) {
        return;
    }
    ow /= op.stride;
    if (ow >= op.w_out) {
        return;
    }

    size_t image_i = b * (op.chan_out * op.h_out * op.w_out) + o * (op.h_out * op.w_out) + oh * (op.w_out)  + ow;
    patches[i] = image_out[image_i];
}

extern "C" __global__ void transpose_and_broadcast_filters(
    const Conv2DOp op,
    const float *filters, // 4d (ChanOut, ChanIn, KernelSize, KernelSize)
    float *filters_tr // 5d (Batch, ChanIn, ChanOut, KernelSize, KernelSize)
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    auto numel = op.chan_in * op.chan_out * op.kernel * op.kernel;
    if (i >= numel) {
        return;
    }

    unsigned int idx = i;
    const size_t k2 = idx % op.kernel;
    idx /= op.kernel;
    const size_t k1 = idx % op.kernel;
    idx /= op.kernel;
    const size_t c = idx % op.chan_in;
    idx /= op.chan_in;
    const size_t o = idx % op.chan_out;
    idx /= op.chan_out;

    auto i_tr = c * (op.chan_out * op.kernel * op.kernel) + o * (op.kernel * op.kernel) + k1 * (op.kernel) + k2;

    const float f = filters[i];
    for (auto b = 0; b < op.batch; b++) {
        filters_tr[b * numel + i_tr] = f;
    }
}

extern "C" __global__ void sum_transposed_filters(
    const Conv2DOp op,
    const float *filters_tr, // 5d (Batch, ChanIn, ChanOut, KernelSize, KernelSize)
    float *filters // 4d (ChanOut, ChanIn, KernelSize, KernelSize)
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    auto numel = op.chan_out * op.chan_in * op.kernel * op.kernel;
    if (i >= numel) {
        return;
    }

    unsigned int idx = i;
    const size_t k2 = idx % op.kernel;
    idx /= op.kernel;
    const size_t k1 = idx % op.kernel;
    idx /= op.kernel;
    const size_t c = idx % op.chan_in;
    idx /= op.chan_in;
    const size_t o = idx % op.chan_out;
    idx /= op.chan_out;

    auto i_tr = c * (op.chan_out * op.kernel * op.kernel) + o * (op.kernel * op.kernel) + k1 * (op.kernel) + k2;

    float tmp = 0.0;
    for (auto b = 0; b < op.batch; b++) {
        tmp += filters_tr[b * numel + i_tr];
    }

    filters[i] += tmp;
}